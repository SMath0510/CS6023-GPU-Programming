#include<iostream>
#include<hip/hip_runtime.h>

__global__
void increment(int * x){
    atomicAdd(x, 1);
    // x[0] += 1;
}

int main(){
    int * x_g, * x_c;
    x_c = (int *) malloc(sizeof(int));
    hipMalloc(&x_g, sizeof(int));
    // increment<<<1,100>>> (x_g); -> 1
    increment<<<100,1>>> (x_g); // -> 1
    hipMemcpy(x_c, x_g, sizeof(int), hipMemcpyDeviceToHost);
    printf("%d\n", x_c[0]);

}