#include <iostream>
#include <hip/hip_runtime.h>

// Define a struct representing a 2D point
struct Point {
    float x;
    float y;
};

// Kernel function to modify the coordinates of a point
__global__ void modifyPoint(Point* point) {
    // Modify the coordinates of the point in the kernel
    point->x *= 2;
    point->y *= 2;
}

int main() {
    // Declare and initialize a point on the host (CPU)
    Point hostPoint = {1.0f, 2.0f};

    // Allocate memory for the point on the GPU
    Point* devicePoint;
    hipMalloc(&devicePoint, sizeof(Point));

    // Copy the point from host to device
    hipMemcpy(devicePoint, &hostPoint, sizeof(Point), hipMemcpyHostToDevice);

    // Launch the kernel to modify the coordinates of the point
    modifyPoint<<<1, 1>>>(devicePoint);
    hipDeviceSynchronize();

    // Copy the modified point back from device to host
    hipMemcpy(&hostPoint, devicePoint, sizeof(Point), hipMemcpyDeviceToHost);

    // Print the modified point
    printf("Modified Point: (%.2f, %.2f)\n", hostPoint.x, hostPoint.y);

    // Free memory on the device
    hipFree(devicePoint);

    return 0;
}
