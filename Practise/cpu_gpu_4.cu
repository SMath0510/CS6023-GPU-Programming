#include<stdio.h>
#include<hip/hip_runtime.h>

__device__ int shaun;

__host__ __device__ 
void fun(int * count){printf("%d\n", *count);} 

__global__
void check(int * ccount, int *gcount){
    fun(ccount);  // prints nothing, just empty's buffer
    *gcount = 100; 
}

int main(){
    int * gcount;
    hipMalloc(&gcount, sizeof(int));
    int * ccount = (int *)malloc(sizeof(int));
    hipMemset(gcount, 0, sizeof(int));
    memset(ccount, 0, sizeof(int));

    *ccount = 10;
    check<<<1,1>>> (ccount, gcount); 
    hipDeviceSynchronize();
    // fun(gcount); // Gives seg fault
}